#include "hip/hip_runtime.h"
#define BOOST_TEST_MODULE gridmaker_cuda_test
#include <boost/test/unit_test.hpp>
#include "grid_maker.h"
#include "atom_typer.h"
#include "test_util.h"
#include <iostream>
#include <iomanip>

#define TOL 0.0001f
using namespace libmolgrid;

BOOST_AUTO_TEST_CASE(forward_agreement) {
  size_t natoms = 1000;
  float resolution = 0.5; 
  float dimension = 23.5;
  float radiusmultiple = 1.5;
  GridMaker gmaker(resolution, dimension, radiusmultiple);
  float3 dim = gmaker.getGridDims();

  //randomly generated example, check equivalence between gpu and cpu versions
  random_engine.seed(0);
  MGrid2f coords(natoms, 3);
  MGrid1f type_indices(natoms); 
  MGrid1f radii(natoms);
  MGrid4f cout(dim.x, dim.y, dim.z, GninaIndexTyper::NumTypes);
  make_mol(coords.cpu(), type_indices.cpu(), radii.cpu(), natoms);
  float3 grid_center = make_float3(0,0,0); //coords generated from -25 to 25
                                           //so this should be ok

  //make grid
  std::fill(cout.data(), cout.data() + cout.size(), 0.0);
  gmaker.forward(grid_center, coords.cpu(), type_indices.cpu(), radii.cpu(), cout.cpu());

  Grid2fCUDA gcoords = coords.gpu();
  Grid1fCUDA gtype_indices = type_indices.gpu();
  Grid1fCUDA gradii = radii.gpu();
  size_t ntypes = GninaIndexTyper::NumTypes;
  size_t gsize = dim.x * dim.y * dim.z * ntypes;
  MGrid4f gout(dim.x, dim.y, dim.z, ntypes);
  LMG_CUDA_CHECK(hipMemset(gout.data(), 0, gsize * sizeof(float)));
  gmaker.forward(grid_center, gcoords, gtype_indices, gradii, gout.gpu());
  hipError_t error = hipGetLastError();
  BOOST_CHECK_EQUAL(error, hipSuccess);
  gout.tocpu();

  // std::ofstream out("out");
  // out.precision(5);
  // std::setprecision(5);
  //check equivalence
  for (size_t i=0; i<dim.x; ++i) {
    for (size_t j=0; j<dim.y; ++j) {
      for (size_t k=0; k<dim.z; ++k) {
        for (size_t ch=0; ch<GninaIndexTyper::NumTypes; ++ch) {
          // out << cout(i,j,k,ch);
          // out << " ";
          // out << gout(i,j,k,ch);
          // out << "\n";
          BOOST_CHECK_SMALL(cout(i,j,k,ch) - gout(i,j,k,ch), TOL);
        }
      }
    }
  }

  //check grid wasn't empty
  BOOST_CHECK_EQUAL(grid_empty(cout.cpu()), false);
  BOOST_CHECK_EQUAL(grid_empty(gout.cpu()), false);
}
