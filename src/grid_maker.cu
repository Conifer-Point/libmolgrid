#include "hip/hip_runtime.h"
#include "grid_maker.h"

namespace libmolgrid {
    __shared__ uint scanScratch[LMG_CUDA_NUM_THREADS * 2];
    __shared__ uint scanBuffer[2][LMG_CUDA_NUM_THREADS];
    __shared__ uint scanOutput[LMG_CUDA_NUM_THREADS];
    __shared__ uint atomIndices[LMG_CUDA_NUM_THREADS];
    __shared__ uint atomMask[LMG_CUDA_NUM_THREADS];

    template <typename Dtype>
    __device__ void zero_grid(Grid<Dtype, 4, true>& grid) {
      size_t gsize = grid.size();
      Dtype* gdata = grid.data();
      size_t bIdx = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
      size_t tidx = bIdx * (blockDim.x * blockDim.y * blockDim.z)
                      + (threadIdx.z * (blockDim.x * blockDim.y))
                      + (threadIdx.y * blockDim.x) + threadIdx.x;
      if (tidx < gsize) 
        gdata[tidx] = 0;
    }

    //TODO: warp shuffle version
    inline __device__ uint warpScanInclusive(int threadIndex, uint idata,
        volatile uint *s_Data, uint size) {
      uint pos = 2 * threadIndex - (threadIndex & (size - 1));
      s_Data[pos] = 0;
      pos += size;
      s_Data[pos] = idata;
    
      for (uint offset = 1; offset < size; offset <<= 1)
        s_Data[pos] += s_Data[pos - offset];
    
      return s_Data[pos];
    }
    
    inline __device__ uint warpScanExclusive(int threadIndex, uint idata,
        volatile uint *sScratch, uint size) {
      return warpScanInclusive(threadIndex, idata, sScratch, size) - idata;
    }
    
    __inline__ __device__ void sharedMemExclusiveScan(int threadIndex, uint* sInput,
        uint* sOutput) {
      uint idata = sInput[threadIndex];
      //Bottom-level inclusive warp scan
      uint warpResult = warpScanInclusive(threadIndex, idata, scanScratch,
          WARP_SIZE);
    
      // Save top elements of each warp for exclusive warp scan sync
      // to wait for warp scans to complete (because s_Data is being
      // overwritten)
      __syncthreads();
    
      if ((threadIndex & (WARP_SIZE - 1)) == (WARP_SIZE - 1)) {
        scanScratch[threadIndex >> LOG2_WARP_SIZE] = warpResult;
      }
    
      // wait for warp scans to complete
      __syncthreads();
    
      if (threadIndex < (LMG_CUDA_NUM_THREADS / WARP_SIZE)) {
        // grab top warp elements
        uint val = scanScratch[threadIndex];
        // calculate exclusive scan and write back to shared memory
        scanScratch[threadIndex] = warpScanExclusive(threadIndex, val, scanScratch,
            LMG_CUDA_NUM_THREADS >> LOG2_WARP_SIZE);
      }
    
      //return updated warp scans with exclusive scan results
      __syncthreads();
    
      sOutput[threadIndex] = warpResult + scanScratch[threadIndex >> LOG2_WARP_SIZE]
          - idata;
    }
    
    //return 1 if atom potentially overlaps block, 0 otherwise
    __device__
    unsigned GridMaker::atomOverlapsBlock(unsigned aidx, float3& grid_origin, 
        const Grid<float, 2, true>& coords, const Grid<float, 1, true>& type_index, 
        const Grid<float, 1, true>& radii) {
   
      if (type_index(aidx) < 0) return 0; //hydrogen
    
      unsigned xi = blockIdx.x * blockDim.x;
      unsigned yi = blockIdx.y * blockDim.y;
      unsigned zi = blockIdx.z * blockDim.z;
    
      //compute corners of block
      float startx = xi * resolution + grid_origin.x;
      float starty = yi * resolution + grid_origin.y;
      float startz = zi * resolution + grid_origin.z;
    
      float endx = startx + resolution * blockDim.x;
      float endy = starty + resolution * blockDim.y;
      float endz = startz + resolution * blockDim.z;
    
      float r = radii(aidx) * radiusmultiple;
      float centerx = coords(aidx, 0);
      float centery = coords(aidx, 1);
      float centerz = coords(aidx, 2);
    
      //does atom overlap box?
      return !((centerx - r > endx) || (centerx + r < startx)
          || (centery - r > endy) || (centery + r < starty)
          || (centerz - r > endz) || (centerz + r < startz));
    }

    template <typename Dtype>
    __device__ void GridMaker::set_atoms(size_t rel_atoms, float3& grid_origin, 
        const Grid<float, 2, true>& coords, const Grid<float, 1, true>& type_index, 
        const Grid<float, 1, true>& radii, Grid<Dtype, 4, true>& out) {
      //figure out what grid point we are 
      uint3 grid_indices;
      grid_indices.x = threadIdx.x + blockIdx.x * blockDim.x;
      grid_indices.y = threadIdx.y + blockIdx.y * blockDim.y;
      grid_indices.z = threadIdx.z + blockIdx.z * blockDim.z;

      if(grid_indices.x >= dim || grid_indices.y >= dim || grid_indices.z >= dim)
        return;//bail if we're off-grid, this should not be common

      //compute x,y,z coordinate of grid point
      float3 grid_coords;
      grid_coords.x = grid_indices.x * resolution + grid_origin.x;
      grid_coords.y = grid_indices.y * resolution + grid_origin.y;
      grid_coords.z = grid_indices.z * resolution + grid_origin.z;

      //iterate over all possibly relevant atoms
      for(size_t ai = 0; ai < rel_atoms; ai++) {
        size_t i = atomIndices[ai];
        float atype = type_index(i);
        if (atype >= 0) { //because of hydrogens on ligands
          float3 acoords;
          acoords.x = coords(i, 0);
          acoords.y = coords(i, 1);
          acoords.z = coords(i, 2);
          float ar = radii(i);
          float val = calcPoint(acoords, ar, grid_coords);
            if(binary) {
              if(val != 0) {
                out(atype, grid_indices.x, grid_indices.y, grid_indices.z) = 1.0;
              }
            }
            else {
                out(atype, grid_indices.x, grid_indices.y, grid_indices.z) += val;
            }
        }
      }
    }

    template <typename Dtype>
    __global__ void forward_gpu(GridMaker gmaker, float3 grid_origin,
        const Grid<float, 2, true> coords, const Grid<float, 1, true> type_index, 
        const Grid<float, 1, true> radii, Grid<Dtype, 4, true> out) {
      //zero out memory buffer
      zero_grid(out);

      //this is the thread's index within its block, used to parallelize over atoms
      size_t total_atoms = coords.dimension(0);
      size_t tidx = ((threadIdx.z * blockDim.z) + threadIdx.y) * blockDim.y + threadIdx.x;
      //if there are more then LMG_CUDA_NUM_THREADS atoms, chunk them
      for(size_t atomoffset = 0; atomoffset < total_atoms; atomoffset += LMG_CUDA_NUM_THREADS) {
        //first parallelize over atoms to figure out if they might overlap this block
        size_t aidx = atomoffset + tidx;
        
        if(aidx < total_atoms) {
          atomMask[tidx] = gmaker.atomOverlapsBlock(aidx, grid_origin, coords,
              type_index, radii);
        }
        else {
          atomMask[tidx] = 0;
        }

        __syncthreads();
        
        //scan the mask to get just relevant indices
        sharedMemExclusiveScan(tidx, atomMask, scanOutput);
        
        __syncthreads();
        
        //do scatter (stream compaction)
        if(atomMask[tidx])
        {
          atomIndices[scanOutput[tidx]] = tidx + atomoffset;
        }
        __syncthreads();

        size_t rel_atoms = scanOutput[LMG_CUDA_NUM_THREADS - 1] + atomMask[LMG_CUDA_NUM_THREADS - 1];
        //atomIndex is now a list of rel_atoms possibly relevant atom indices
        gmaker.set_atoms(rel_atoms, grid_origin, coords, type_index, radii, out);
        __syncthreads();//everyone needs to finish before we muck with atomIndices again
      }
    }

    template <typename Dtype>
    void GridMaker::forward(float3 grid_center, const Grid<float, 2, true>& coords,
        const Grid<float, 1, true>& type_index, const Grid<float, 1, true>& radii,
        Grid<Dtype, 4, true>& out) const {
      //threads are laid out in three dimensions to match the voxel grid, 
      //8x8x8=512 threads per block
      dim3 threads(LMG_CUDA_BLOCKDIM, LMG_CUDA_BLOCKDIM, LMG_CUDA_BLOCKDIM);
      unsigned blocksperside = ceil(dim / float(LMG_CUDA_BLOCKDIM));
      dim3 blocks(blocksperside, blocksperside, blocksperside);
      float3 grid_origin = getGridOrigin(grid_center);
      forward_gpu<Dtype><<<blocks, threads>>>(*this, grid_origin, coords, type_index, radii, out);
      LMG_CUDA_CHECK(hipPeekAtLastError());
    }

    template 
    void GridMaker::forward(float3 grid_center, const Grid<float, 2, true>& coords,
        const Grid<float, 1, true>& type_index, const Grid<float, 1, true>& radii,
        Grid<float, 4, true>& out) const;
} /* namespace libmolgrid */
