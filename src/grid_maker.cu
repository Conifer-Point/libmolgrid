#include "hip/hip_runtime.h"
#include "libmolgrid/grid_maker.h"

namespace libmolgrid {
    __shared__ uint scanScratch[LMG_CUDA_NUM_THREADS * 2];
    __shared__ uint scanBuffer[2][LMG_CUDA_NUM_THREADS];
    __shared__ uint scanOutput[LMG_CUDA_NUM_THREADS];
    __shared__ uint atomIndices[LMG_CUDA_NUM_THREADS];
    __shared__ uint atomMask[LMG_CUDA_NUM_THREADS];

    template <typename Dtype>
    __device__ void zero_grid(Grid<Dtype, 4, true>& grid) {
      size_t gsize = grid.size();
      Dtype* gdata = grid.data();
      size_t bIdx = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
      size_t tidx = bIdx * (blockDim.x * blockDim.y * blockDim.z)
                      + (threadIdx.z * (blockDim.x * blockDim.y))
                      + (threadIdx.y * blockDim.x) + threadIdx.x;
      if (tidx < gsize) 
        gdata[tidx] = 0;
    }

    template __device__ void zero_grid(Grid<float, 4, true> & grid);

    //TODO: warp shuffle version
    inline __device__ uint warpScanInclusive(int threadIndex, uint idata,
        volatile uint *s_Data, uint size) {
      uint pos = 2 * threadIndex - (threadIndex & (size - 1));
      s_Data[pos] = 0;
      pos += size;
      s_Data[pos] = idata;
    
      for (uint offset = 1; offset < size; offset <<= 1)
        s_Data[pos] += s_Data[pos - offset];
    
      return s_Data[pos];
    }
    
    inline __device__ uint warpScanExclusive(int threadIndex, uint idata,
        volatile uint *sScratch, uint size) {
      return warpScanInclusive(threadIndex, idata, sScratch, size) - idata;
    }
    
    __inline__ __device__ void sharedMemExclusiveScan(int threadIndex, uint* sInput,
        uint* sOutput) {
      uint idata = sInput[threadIndex];
      //Bottom-level inclusive warp scan
      uint warpResult = warpScanInclusive(threadIndex, idata, scanScratch,
          WARP_SIZE);
    
      // Save top elements of each warp for exclusive warp scan sync
      // to wait for warp scans to complete (because s_Data is being
      // overwritten)
      __syncthreads();
    
      if ((threadIndex & (WARP_SIZE - 1)) == (WARP_SIZE - 1)) {
        scanScratch[threadIndex >> LOG2_WARP_SIZE] = warpResult;
      }
    
      // wait for warp scans to complete
      __syncthreads();
    
      if (threadIndex < (LMG_CUDA_NUM_THREADS / WARP_SIZE)) {
        // grab top warp elements
        uint val = scanScratch[threadIndex];
        // calculate exclusive scan and write back to shared memory
        scanScratch[threadIndex] = warpScanExclusive(threadIndex, val, scanScratch,
            LMG_CUDA_NUM_THREADS >> LOG2_WARP_SIZE);
      }
    
      //return updated warp scans with exclusive scan results
      __syncthreads();
    
      sOutput[threadIndex] = warpResult + scanScratch[threadIndex >> LOG2_WARP_SIZE]
          - idata;
    }
    
    //return 1 if atom potentially overlaps block, 0 otherwise
    __device__
    unsigned GridMaker::atom_overlaps_block(unsigned aidx, float3& grid_origin, 
        const Grid<float, 2, true>& coords, const Grid<float, 1, true>& type_index, 
        const Grid<float, 1, true>& radii) {
   
      if (type_index(aidx) < 0) return 0; //hydrogen
    
      unsigned xi = blockIdx.x * blockDim.x;
      unsigned yi = blockIdx.y * blockDim.y;
      unsigned zi = blockIdx.z * blockDim.z;
    
      //compute corners of block
      float startx = xi * resolution + grid_origin.x;
      float starty = yi * resolution + grid_origin.y;
      float startz = zi * resolution + grid_origin.z;
    
      float endx = startx + resolution * blockDim.x;
      float endy = starty + resolution * blockDim.y;
      float endz = startz + resolution * blockDim.z;
    
      float r = radii(aidx) * radiusmultiple;
      float centerx = coords(aidx, 0);
      float centery = coords(aidx, 1);
      float centerz = coords(aidx, 2);
    
      //does atom overlap box?
      return !((centerx - r > endx) || (centerx + r < startx)
          || (centery - r > endy) || (centery + r < starty)
          || (centerz - r > endz) || (centerz + r < startz));
    }

    template <typename Dtype>
    __device__ void GridMaker::set_atoms(size_t rel_atoms, float3& grid_origin, 
        const Grid<float, 2, true>& coords, const Grid<float, 1, true>& type_index, 
        const Grid<float, 1, true>& radii, Grid<Dtype, 4, true>& out) {
      //figure out what grid point we are 
      uint3 grid_indices;
      grid_indices.x = threadIdx.x + blockIdx.x * blockDim.x;
      grid_indices.y = threadIdx.y + blockIdx.y * blockDim.y;
      grid_indices.z = threadIdx.z + blockIdx.z * blockDim.z;

      if(grid_indices.x >= dim || grid_indices.y >= dim || grid_indices.z >= dim)
        return;//bail if we're off-grid, this should not be common

      size_t ntypes = out.dimension(0);
      //compute x,y,z coordinate of grid point
      float3 grid_coords;
      grid_coords.x = grid_indices.x * resolution + grid_origin.x;
      grid_coords.y = grid_indices.y * resolution + grid_origin.y;
      grid_coords.z = grid_indices.z * resolution + grid_origin.z;

      //iterate over all possibly relevant atoms
      for(size_t ai = 0; ai < rel_atoms; ai++) {
        size_t i = atomIndices[ai];
        float atype = type_index(i);
        if (atype >= 0 && atype < ntypes) { //should really throw an exception here, but can't
          float3 acoords;
          acoords.x = coords(i, 0);
          acoords.y = coords(i, 1);
          acoords.z = coords(i, 2);
          float ar = radii(i);
          float val = calc_point(acoords, ar, grid_coords);
            if(binary) {
              if(val != 0) {
                out(atype, grid_indices.x, grid_indices.y, grid_indices.z) = 1.0;
              }
            }
            else {
                // out(atype, grid_indices.x, grid_indices.y, grid_indices.z) += val;
              size_t offset = ((((atype * dim) + grid_indices.x) * dim) +
                  grid_indices.y) * dim + grid_indices.z;
              *(out.data() + offset) += val;
            }
        }
      }
    }

    template <typename Dtype>
    __global__ void forward_gpu(GridMaker gmaker, float3 grid_origin,
        const Grid<float, 2, true> coords, const Grid<float, 1, true> type_index, 
        const Grid<float, 1, true> radii, Grid<Dtype, 4, true> out) {
      //this is the thread's index within its block, used to parallelize over atoms
      size_t total_atoms = coords.dimension(0);
      size_t tidx = ((threadIdx.z * blockDim.y) + threadIdx.y) * blockDim.x + threadIdx.x;
      //if there are more then LMG_CUDA_NUM_THREADS atoms, chunk them
      for(size_t atomoffset = 0; atomoffset < total_atoms; atomoffset += LMG_CUDA_NUM_THREADS) {
        //first parallelize over atoms to figure out if they might overlap this block
        size_t aidx = atomoffset + tidx;
        
        if(aidx < total_atoms) {
          atomMask[tidx] = gmaker.atom_overlaps_block(aidx, grid_origin, coords,
              type_index, radii);
        }
        else {
          atomMask[tidx] = 0;
        }

        __syncthreads();
        
        //scan the mask to get just relevant indices
        sharedMemExclusiveScan(tidx, atomMask, scanOutput);
        
        __syncthreads();
        
        //do scatter (stream compaction)
        if(atomMask[tidx])
        {
          atomIndices[scanOutput[tidx]] = tidx + atomoffset;
        }
        __syncthreads();

        size_t rel_atoms = scanOutput[LMG_CUDA_NUM_THREADS - 1] + atomMask[LMG_CUDA_NUM_THREADS - 1];
        //atomIndex is now a list of rel_atoms possibly relevant atom indices
        gmaker.set_atoms(rel_atoms, grid_origin, coords, type_index, radii, out);
        __syncthreads();//everyone needs to finish before we muck with atomIndices again
      }
    }

    template <typename Dtype>
    void GridMaker::forward(float3 grid_center, const Grid<float, 2, true>& coords,
        const Grid<float, 1, true>& type_index, const Grid<float, 1, true>& radii,
        Grid<Dtype, 4, true>& out) const {
      //threads are laid out in three dimensions to match the voxel grid, 
      //8x8x8=512 threads per block
      dim3 threads(LMG_CUDA_BLOCKDIM, LMG_CUDA_BLOCKDIM, LMG_CUDA_BLOCKDIM);
      unsigned blocksperside = ceil(dim / float(LMG_CUDA_BLOCKDIM));
      dim3 blocks(blocksperside, blocksperside, blocksperside);
      float3 grid_origin = get_grid_origin(grid_center);

      check_index_args(coords, type_index, radii, out);
      //zero out grid to start
      LMG_CUDA_CHECK(hipMemset(out.data(), 0, out.size() * sizeof(float)));
      forward_gpu<Dtype><<<blocks, threads>>>(*this, grid_origin, coords, type_index, radii, out);
      LMG_CUDA_CHECK(hipPeekAtLastError());
    }

    template void GridMaker::forward(float3 grid_center, const Grid<float, 2, true>& coords,
        const Grid<float, 1, true>& type_index, const Grid<float, 1, true>& radii,
        Grid<float, 4, true>& out) const;
    template void GridMaker::forward(float3 grid_center, const Grid<float, 2, true>& coords,
        const Grid<float, 1, true>& type_index, const Grid<float, 1, true>& radii,
        Grid<double, 4, true>& out) const;


    float GridMaker::calc_point(const float3& coords, double ar,
            const float3& grid_coords) const {
          float dx = grid_coords.x - coords.x;
          float dy = grid_coords.y - coords.y;
          float dz = grid_coords.z - coords.z;

          float rsq = dx * dx + dy * dy + dz * dz;
          if (binary) {
            //is point within radius?
            if (rsq < ar * ar)
              return 1.0;
            else
              return 0.0;
          } else {
            //For non-binary density we want a Gaussian where 2 std occurs at the
            //radius, after which it becomes quadratic.
            //The quadratic is fit to have both the same value and first derivative
            //at the cross over point and a value and derivative of zero at
            //1.5*radius
            //FIXME wrong for radiusmultiple != 1.5
            float dist = sqrtf(rsq);
            if (dist >= ar * radiusmultiple) {
              return 0.0;
            } else
              if (dist <= ar) {
                //return gaussian
                float h = 0.5 * ar;
                float ex = -dist * dist / (2 * h * h);
                return exp(ex);
              } else //return quadratic
              {
                float h = 0.5 * ar;
                float eval = 1.0 / (M_E * M_E); //e^(-2)
                float q = dist * dist * eval / (h * h) - 6.0 * eval * dist / h
                    + 9.0 * eval;
                return q > 0 ? q : 0; //avoid very small negative numbers
              }
          }
        }



} /* namespace libmolgrid */
