#include "hip/hip_runtime.h"
#include "grid_maker.h"

namespace libmolgrid {
    __shared__ uint scanScratch[LMG_CUDA_NUM_THREADS * 2];
    __shared__ uint scanBuffer[2][LMG_CUDA_NUM_THREADS];
    __shared__ uint scanOutput[LMG_CUDA_NUM_THREADS];
    __shared__ uint atomIndices[LMG_CUDA_NUM_THREADS];
    __shared__ uint atomMask[LMG_CUDA_NUM_THREADS];

    template <typename Dtype>
    __device__ void zero_grid(Grid<Dtype, 4, true> grid) {
      size_t gsize = grid.size();
      Dtype* gdata = grid.data();
      size_t bIdx = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
      size_t tidx = bIdx * (blockDim.x * blockDim.y * blockDim.z)
                      + (threadIdx.z * (blockDim.x * blockDim.y))
                      + (threadIdx.y * blockDim.x) + threadIdx.x;
      if (tidx < gsize) 
        gdata[tidx] = 0;
    }

    //return squared distance between pt and (x,y,z)
    __device__
    float sqDistance(float coordx, float coordy, float coordz, float x, float y, float z) {
      float ret;
      float tmp = coordx - x;
      ret = tmp * tmp;
      tmp = coordy - y;
      ret += tmp * tmp;
      tmp = coordz - z;
      ret += tmp * tmp;
      return ret;
    }

    //TODO: warp shuffle version
    inline __device__ uint warpScanInclusive(int threadIndex, uint idata,
        volatile uint *s_Data, uint size) {
      uint pos = 2 * threadIndex - (threadIndex & (size - 1));
      s_Data[pos] = 0;
      pos += size;
      s_Data[pos] = idata;
    
      for (uint offset = 1; offset < size; offset <<= 1)
        s_Data[pos] += s_Data[pos - offset];
    
      return s_Data[pos];
    }
    
    inline __device__ uint warpScanExclusive(int threadIndex, uint idata,
        volatile uint *sScratch, uint size) {
      return warpScanInclusive(threadIndex, idata, sScratch, size) - idata;
    }
    
    __inline__ __device__ void sharedMemExclusiveScan(int threadIndex, uint* sInput,
        uint* sOutput) {
      uint idata = sInput[threadIndex];
      //Bottom-level inclusive warp scan
      uint warpResult = warpScanInclusive(threadIndex, idata, scanScratch,
          WARP_SIZE);
    
      // Save top elements of each warp for exclusive warp scan sync
      // to wait for warp scans to complete (because s_Data is being
      // overwritten)
      __syncthreads();
    
      if ((threadIndex & (WARP_SIZE - 1)) == (WARP_SIZE - 1)) {
        scanScratch[threadIndex >> LOG2_WARP_SIZE] = warpResult;
      }
    
      // wait for warp scans to complete
      __syncthreads();
    
      if (threadIndex < (LMG_CUDA_NUM_THREADS / WARP_SIZE)) {
        // grab top warp elements
        uint val = scanScratch[threadIndex];
        // calculate exclusive scan and write back to shared memory
        scanScratch[threadIndex] = warpScanExclusive(threadIndex, val, scanScratch,
            LMG_CUDA_NUM_THREADS >> LOG2_WARP_SIZE);
      }
    
      //return updated warp scans with exclusive scan results
      __syncthreads();
    
      sOutput[threadIndex] = warpResult + scanScratch[threadIndex >> LOG2_WARP_SIZE]
          - idata;
    }
    
    //return 1 if atom potentially overlaps block, 0 otherwise
    __device__
    unsigned GridMaker::atomOverlapsBlock(unsigned aidx, float3 grid_origin, 
        const Grid<float, 2, true>& coords, const Grid<float, 1, true>& type_index, 
        const Grid<float, 1, true>& radii) {
   
      //FIXME: include hydrogens?
      if (type_index(aidx) < 0) return 0; //hydrogen
    
      unsigned xi = blockIdx.x * blockDim.x;
      unsigned yi = blockIdx.y * blockDim.y;
      unsigned zi = blockIdx.z * blockDim.z;
    
      //compute corners of block
      float startx = xi * resolution + grid_origin.x;
      float starty = yi * resolution + grid_origin.y;
      float startz = zi * resolution + grid_origin.z;
    
      float endx = startx + resolution * blockDim.x;
      float endy = starty + resolution * blockDim.y;
      float endz = startz + resolution * blockDim.z;
    
      float r = radii(aidx) * radiusmultiple;
      float centerx = coords(aidx, 0);
      float centery = coords(aidx, 1);
      float centerz = coords(aidx, 2);
    
      //does atom overlap box?
      return !((centerx - r > endx) || (centerx + r < startx)
          || (centery - r > endy) || (centery + r < starty)
          || (centerz - r > endz) || (centerz + r < startz));
    }

    template <typename Dtype>
    __device__ void GridMaker::set_atoms(unsigned rel_atoms, float3& grid_origin, 
        const Grid<float, 2, true>& coords, const Grid<float, 1, true>& type_index, 
        const Grid<float, 1, true>& radii, Grid<Dtype, 4, true>& out) {
      //figure out what grid point we are 
      unsigned xi = threadIdx.x + blockIdx.x * blockDim.x;
      unsigned yi = threadIdx.y + blockIdx.y * blockDim.y;
      unsigned zi = threadIdx.z + blockIdx.z * blockDim.z;

      if(xi >= dim || yi >= dim || zi >= dim)
        return;//bail if we're off-grid, this should not be common

      //compute x,y,z coordinate of grid point
      float x = xi * resolution + grid_origin.x;
      float y = yi * resolution + grid_origin.y;
      float z = zi * resolution + grid_origin.z;

      //iterate over all atoms
      for(unsigned ai = 0; ai < rel_atoms; ai++) {
        unsigned i = atomIndices[ai];
        float coordx = coords(i, 0);
        float coordy = coords(i, 1);
        float coordz = coords(i, 2);
        float d = sqDistance(coordx, coordy, coordz, x, y, z);
        float r = radii(i);
        float atype = type_index(i);
        if (atype >= 0) {    //because of hydrogens on ligands, although 
                             //we already excluded them from the atom list...
          float rsq = r * r;

          if(binary) {
            if(d < rsq) {
              //set gridpoint to 1
              out(atype, xi, yi, zi) = 1.0;
            }
          }
          else {
            //For non-binary density we want a Gaussian where 2 std occurs at the
            //radius, after which it becomes quadratic.  
            //The quadratic is fit to have both the same value and first derivative
            //at the cross over point and a value and derivative of zero at
            //1.5*radius 
            //FIXME wrong for radiusmultiple != 1.5
            float dist = sqrtf(d);
            if (dist < r * radiusmultiple) {
              float h = 0.5 * r;

              if (dist <= r) {
                //return gaussian
                float ex = -dist * dist / (2 * h * h);
                out(atype, xi, yi, zi) += exp(ex);
              }
              else {//return quadratic
                float eval = 1.0 / (M_E * M_E); //e^(-2)
                float q = dist * dist * eval / (h * h) - 6.0 * eval * dist / h + 9.0 * eval;
                out(atype, xi, yi, zi) += q;
              }
            }
          }
        }
      }
    }

    template <typename Dtype>
    __global__ void forward_gpu(GridMaker gmaker, float3 grid_origin,
        const Grid<float, 2, true> coords, const Grid<float, 1, true> type_index, 
        const Grid<float, 1, true> radii, Grid<Dtype, 4, true> out) {
      //zero out memory buffer
      zero_grid(out);

      //this is the thread's index within its block, used to parallelize over atoms
      unsigned total_atoms = radii.size();
      size_t tidx = ((threadIdx.z * blockDim.z) + threadIdx.y) * blockDim.y + threadIdx.x;
      //if there are more then LMG_CUDA_NUM_THREADS atoms, chunk them
      for(unsigned atomoffset = 0; atomoffset < total_atoms; atomoffset += LMG_CUDA_NUM_THREADS) {
        //first parallelize over atoms to figure out if they might overlap this block
        unsigned aidx = atomoffset + tidx;
        
        if(aidx < total_atoms) {
          atomMask[tidx] = gmaker.atomOverlapsBlock(aidx, grid_origin, coords,
              type_index, radii);
        }
        else {
          atomMask[tidx] = 0;
        }

        __syncthreads();
        
        //scan the mask to get just relevant indices
        sharedMemExclusiveScan(tidx, atomMask, scanOutput);
        
        __syncthreads();
        
        //do scatter (stream compaction)
        if(atomMask[tidx])
        {
          atomIndices[scanOutput[tidx]] = tidx + atomoffset;
        }
        __syncthreads();

        unsigned rel_atoms = scanOutput[LMG_CUDA_NUM_THREADS - 1] + atomMask[LMG_CUDA_NUM_THREADS - 1];
        //atomIndex is now a list of rel_atoms possibly relevant atom indices
        gmaker.set_atoms(rel_atoms, grid_origin, coords, type_index, radii, out);
        __syncthreads();//everyone needs to finish before we muck with atomIndices again
      }
    }

    template <typename Dtype>
    void GridMaker::forward(float3 grid_center, const Grid<float, 2, true>& coords,
        const Grid<float, 1, true>& type_index, const Grid<float, 1, true>& radii,
        Grid<Dtype, 4, true>& out) const {
      //threads are laid out in three dimensions to match the voxel grid, 
      //8x8x8=512 threads per block
      dim3 threads(LMG_CUDA_BLOCKDIM, LMG_CUDA_BLOCKDIM, LMG_CUDA_BLOCKDIM);
      unsigned blocksperside = ceil(dim / float(LMG_CUDA_BLOCKDIM));
      dim3 blocks(blocksperside, blocksperside, blocksperside);
      float3 grid_origin = getGridOrigin(grid_center);
      forward_gpu<Dtype><<<blocks, threads>>>(*this, grid_origin, coords, type_index, radii, out);
      LMG_CUDA_CHECK(hipPeekAtLastError());
    }
} /* namespace libmolgrid */
